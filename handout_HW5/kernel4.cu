#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE  16

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im; //re: real part,  im: imaginary part
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int maxIterations, int *device_result) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thread_position_x = blockIdx.x * blockDim.x + threadIdx.x; //threadIdx is the threadID in this specific block
    int thread_position_y = blockIdx.y * blockDim.y + threadIdx.y;

    float x = lowerX + (float)(thread_position_x * stepX);
    float y = lowerY + (float)(thread_position_y * stepY);

    int idx = thread_position_x + (thread_position_y * resX);
    device_result[idx] = mandel(x, y, maxIterations);
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{   
    float stepX = (upperX - lowerX) / resX; //mean a stride of X-axis
    float stepY = (upperY - lowerY) / resY;

    /*
        Allocate Storage on Host & Device
    */

    int *host_result, *device_result; 
    int size = resX * resY * sizeof(int);


    host_result = (int *)malloc(size);
    hipMalloc(&device_result, size);

    /*
        Call Kernel Function
    */
    dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 num_blocks(resX / threads_per_block.x, resY / threads_per_block.y);
    mandelKernel<<<num_blocks, threads_per_block>>>(lowerX, lowerY, stepX, stepY, resX, resY, maxIterations, device_result);
    
    hipMemcpy(host_result, device_result, size, hipMemcpyDeviceToHost);
    memcpy(img, host_result, size);

    // Free allocated memory
    free(host_result);
    hipFree(device_result);
}
