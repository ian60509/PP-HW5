#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE  16
#define GROUP_SIZE 2

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im; //re: real part,  im: imaginary part
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int resX, int resY, int maxIterations, int *device_result, size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    //thread_position_x: the position on the 2D thread hierarchy
    int thread_position_x = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE ; //threadIdx is the threadID in this specific block
    int thread_position_y = ( blockIdx.y * blockDim.y + threadIdx.y) * GROUP_SIZE;

    

    for(int i=thread_position_x; i<thread_position_x+GROUP_SIZE; i++){
        if( i>= resX) break;

        for(int j=thread_position_y; j<thread_position_y+GROUP_SIZE; j++){
            if(j>=resY) break;

            float x = lowerX + (float)(i * stepX); //the actual position on the complex plane
            float y = lowerY + (float)(j * stepY);

            int idx = i + (j * resX);
            device_result[idx] = mandel(x, y, maxIterations);
        }
    }

    
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{   
    float stepX = (upperX - lowerX) / resX; //mean a stride of X-axis
    float stepY = (upperY - lowerY) / resY;

    /*
        Allocate Storage on Host & Device
    */

    int *host_result, *device_result; 
    int size = resX * resY * sizeof(int);

    size_t pitch;
    hipHostAlloc(&host_result, size, hipHostMallocDefault);
    hipMallocPitch(&device_result, &pitch, resX*sizeof(int), resY);


    host_result = (int *)malloc(size);
    hipMalloc(&device_result, size);

    /*
        Call Kernel Function
    */
    dim3 threads_per_block(BLOCK_SIZE/GROUP_SIZE, BLOCK_SIZE/GROUP_SIZE);
    dim3 num_blocks(resX / threads_per_block.x, resY / threads_per_block.y);
    mandelKernel<<<num_blocks, threads_per_block>>>(lowerX, lowerY, stepX, stepY, resX, resY, maxIterations, device_result, pitch);
    
    hipMemcpy(host_result, device_result, size, hipMemcpyDeviceToHost);
    memcpy(img, host_result, size);

    // Free allocated memory
    free(host_result);
    hipFree(device_result);
}
